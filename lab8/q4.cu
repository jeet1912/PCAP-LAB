#include "hip/hip_runtime.h"
#include<stdio.h>
#include<hip/hip_runtime.h>
#include<>
#include<string.h>
__global__ void conc(char *a,int len, int n,char *b)
{
  int id=blockIdx.x*blockDim.x+threadIdx.x;
  for(int i=id;i<n*len;i=i+len)
    b[i]=a[id];
}

int main()
{
  int n,len;
  char a[100],b[100];
  printf("Enter str:");
  gets(a);
  printf("Enter n: ");
  scanf("%d",&n);
  len=strlen(a);
  char *d_a,*d_b;
  memset(b,0,100);
  hipMalloc((void**)&d_a,len);
  hipMalloc((void**)&d_b,len*n);
  hipMemcpy(d_a,&a,len,hipMemcpyHostToDevice);
  hipMemcpy(d_b,&b,len*n,hipMemcpyHostToDevice);
  conc<<<1,len>>>(d_a,len,n,d_b);
  hipMemcpy(&b,d_b,len*n,hipMemcpyDeviceToHost);
  printf("%s\n",b);
  hipFree(d_a);
  hipFree(d_b);
}
