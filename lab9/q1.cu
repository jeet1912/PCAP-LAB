#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>

__global__ void q1(int *a, int *res, int m, int n) 
{
	int id=blockIdx.x*blockDim.x+threadIdx.x;
	res[id]=powf(a[id],blockIdx.x+1);
}

int main() {
	int *a, *res, m,n, i,j;
	int *d_a, *d_res;

	printf("Enter value of m: ");scanf("%d",&m);
	printf("Enter value of n: ");scanf("%d",&n);
	int size= sizeof(int)*m*n;
	a=(int*)malloc(size);
	res=(int*)malloc(size);
	printf("ENter input matrix : \n");
	for(i=0; i<m*n;i++)
		scanf("%d",&a[i]);
	hipMalloc((void **)&d_a,size);
	hipMalloc((void **)&d_res,size);

	hipMemcpy(d_a,a,size,hipMemcpyHostToDevice);
	q1<<<m,n>>>(d_a,d_res,m,n);	

	hipMemcpy(res,d_res,size,hipMemcpyDeviceToHost);

	printf("Result: \n");	
	for(i=0; i<m;i++)
	{
		for(j=0; j<n;j++)
		{
			printf("%d\t",res[i*n+j]);
		}
		printf("\n");
	}
	hipFree(d_a);
	hipFree(d_res);
	return 0;
}