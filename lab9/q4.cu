#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>

__global__ void q3(int *a, int *res, int m, int n) 
{
	int rid=threadIdx.y;
	int cid=threadIdx.x;
	if(rid%(m-1)==0 || cid%(n-1)==0 )
		res[rid*m+cid]=a[rid*m+cid];
	else
	{
		int num=a[rid*m+cid];
		int bin=0, rem,i=1;
		while(num!=0)
		{
			rem=num%2;
			num/=2;
			bin+=(rem==0?1:0)*i;
			i*=10;
		}
		res[rid*m+cid]=bin;
	}
}

int main() {
	int *a, *res, m,n, i,j;
	int *d_a, *d_res;

	printf("Enter value of m: ");scanf("%d",&m);
	printf("Enter value of n: ");scanf("%d",&n);
	int size= sizeof(int)*m*n;
	a=(int*)malloc(size);
	res=(int*)malloc(size);
	printf("ENter input matrix : \n");
	for(i=0; i<m*n;i++)
		scanf("%d",&a[i]);
	hipMalloc((void **)&d_a,size);
	hipMalloc((void **)&d_res,size);

	hipMemcpy(d_a,a,size,hipMemcpyHostToDevice);
	dim3 thr(m,n);
	q3<<<1,thr>>>(d_a,d_res,m,n);	

	hipMemcpy(res,d_res,size,hipMemcpyDeviceToHost);

	printf("Result: \n");	
	for(i=0; i<m;i++)
	{
		for(j=0; j<n;j++)
		{
			printf("%d\t",res[i*n+j]);
		}
		printf("\n");
	}
	hipFree(d_a);
	hipFree(d_res);
	return 0;
}