#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>

__global__ void q3a(int *a, int *b,int *c, int wa, int wb) 
{
	int id=threadIdx.x , sum;
	for(int i=0; i< wb; i++)
	{
		sum=0;
		for(int j=0; j< wa;j++)
		{
			sum+=(a[id*wa+j]*b[j*wb+i]);
		}
		c[id*wb+i]=sum;
	}
}

__global__ void q3b(int *a, int *b,int *c, int ha, int wa, int wb) 
{
	int id=threadIdx.x ,sum;
	for(int i=0; i< ha; i++)
	{
		sum=0;
		for(int j=0; j< wa;j++)
		{
			sum+=(a[i*wa+j]*b[j*wb+id]);
		}
		c[i*wb+id]=sum;
	}
}

__global__ void q3c(int *a, int *b,int *c, int wa, int wb) 
{
	int rid=threadIdx.y;
	int cid=threadIdx.x;
	int sum=0;
	for(int i=0; i< wa;i++)
	{
		sum+=a[rid*wa+i]*b[i*wb+cid];
	}
	c[rid*wb+cid]=sum;

}
void display(int *res,int m,int n)
{
	printf("Result: \n");	
	for(int i=0; i<m;i++)
	{
		for(int j=0; j<n;j++)
		{
			printf("%d\t",res[i*n+j]);
		}
		printf("\n");
	}
}

int main() {
	int *a, *b, m,n,q, i;
	int *d_a,*d_b, *d_res;

	printf("Enter value of m: ");scanf("%d",&m);
	printf("Enter value of n: ");scanf("%d",&n);
	printf("Enter value of q: ");scanf("%d",&q);
	int size= sizeof(int)*m*n;
	a=(int*)malloc(size);

	int *resa=(int*)malloc(size);
	int *resb=(int*)malloc(size);
	int *resc=(int*)malloc(size);

	printf("ENter input matrix A (m*n): \n");
	for(i=0; i<m*n;i++)
		scanf("%d",&a[i]);

	b=(int*)malloc(size);
	printf("ENter input matrix B (p*q): \n");
	for(i=0; i<n*q;i++)
		scanf("%d",&b[i]);
	
	hipMalloc((void **)&d_a,size);
	hipMalloc((void **)&d_b,size);
	hipMalloc((void **)&d_res,size);

	hipMemcpy(d_a,a,size,hipMemcpyHostToDevice);
	hipMemcpy(d_b,b,size,hipMemcpyHostToDevice);
	
	// ha= m , wa=n=p wb= q
	//CALL A
	q3a<<<1,m>>>(d_a,d_b,d_res,n,q);	
	hipMemcpy(resa,d_res,size,hipMemcpyDeviceToHost);
	display(resa,m,q);
	//CALL B
	q3b<<<1,q>>>(d_a,d_b,d_res,m,n,q);	
	hipMemcpy(resb,d_res,size,hipMemcpyDeviceToHost);
	display(resb,m,q);
	//CALL C
	dim3 blo(1,1);
	dim3 thr(q,m);
	q3c<<<blo,thr>>>(d_a,d_b,d_res,n,q);	
	hipMemcpy(resc,d_res,size,hipMemcpyDeviceToHost);
	display(resc,m,q);
	

	hipFree(d_a);
	hipFree(d_res);
	return 0;
}