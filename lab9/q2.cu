#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>

__global__ void q2a(int *a, int *b,int *res, int m, int n) 
{
	int id=threadIdx.x;
	for(int i=0; i<n; i++)
		res[id*n+i]=a[id*n+i]+b[id*n+i];

}

__global__ void q2b(int *a, int *b,int *res, int m, int n) 
{
	int id=threadIdx.x;
	for(int i=0; i<n; i++)
		res[i*n+id]=a[i*n+id]+b[i*n+id];
}

__global__ void q2c(int *a, int *b,int *res, int m, int n) 
{
	int id=blockIdx.x*blockDim.x+threadIdx.x;
	res[id]=a[id]+b[id];
}
void display(int *res,int m,int n)
{
	printf("Result: \n");	
	for(int i=0; i<m;i++)
	{
		for(int j=0; j<n;j++)
		{
			printf("%d\t",res[i*n+j]);
		}
		printf("\n");
	}
}

int main() {
	int *a, *b, m,n, i;
	int *d_a,*d_b, *d_res;

	printf("Enter value of m: ");scanf("%d",&m);
	printf("Enter value of n: ");scanf("%d",&n);
	int size= sizeof(int)*m*n;
	a=(int*)malloc(size);

	int *resa=(int*)malloc(size);
	int *resb=(int*)malloc(size);
	int *resc=(int*)malloc(size);

	printf("ENter input matrix A : \n");
	for(i=0; i<m*n;i++)
		scanf("%d",&a[i]);

	b=(int*)malloc(size);
	printf("ENter input matrix B : \n");
	for(i=0; i<m*n;i++)
		scanf("%d",&b[i]);
	
	hipMalloc((void **)&d_a,size);
	hipMalloc((void **)&d_b,size);
	hipMalloc((void **)&d_res,size);

	hipMemcpy(d_a,a,size,hipMemcpyHostToDevice);
	hipMemcpy(d_b,b,size,hipMemcpyHostToDevice);
	
	//CALL A
	q2a<<<1,m>>>(d_a,d_b,d_res,m,n);	
	hipMemcpy(resa,d_res,size,hipMemcpyDeviceToHost);
	display(resa,m,n);
	//CALL B
	q2b<<<1,n>>>(d_a,d_b,d_res,m,n);	
	hipMemcpy(resb,d_res,size,hipMemcpyDeviceToHost);
	display(resb,m,n);
	//CALL C
	q2c<<<m,n>>>(d_a,d_b,d_res,m,n);	
	hipMemcpy(resc,d_res,size,hipMemcpyDeviceToHost);
	display(resc,m,n);
	

	hipFree(d_a);
	hipFree(d_res);
	return 0;
}