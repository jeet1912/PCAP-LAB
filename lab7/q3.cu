#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <time.h>

#define N 12         // size of array

__global__ void linalg(int *a,int *b, int *c, int *scalar) {
	int tid = blockIdx.x *  blockDim.x + threadIdx.x;
		int s = *scalar;
        if(tid < N){
          c[tid] = s * a[tid] + b[tid];
        }
}

int main(int argc, char *argv[])  {
	int T = 10, B = 1;            // threads per block and blocks per grid
	int a[N],b[N],c[N], *scalar;

	scalar = (int*)malloc(sizeof(int));
	int *dev_a, *dev_b, *dev_c, *dev_scalar;

	printf("Enter the scalar:\n");
	scanf("%d", scalar);

	printf("Size of array = %d\n", N);
	do {
		printf("Enter number of threads per block: ");
		scanf("%d",&T);
		printf("\nEnter number of blocks per grid: ");
		scanf("%d",&B);
		if (T * B != N) printf("Error T x B != N, try again");
	} while (T * B != N);

	hipMalloc((void**)&dev_a,N * sizeof(int));
	hipMalloc((void**)&dev_b,N * sizeof(int));
	hipMalloc((void**)&dev_c,N * sizeof(int));
	hipMalloc((void**)&dev_scalar, sizeof(int));

	for(int i=0;i<N;i++) {    // load arrays with some numbers
		a[i] = i;
		b[i] = i*1;
	}

	hipMemcpy(dev_a, a, N*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, N*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_c, c, N*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_scalar, scalar, sizeof(int), hipMemcpyHostToDevice);

	linalg<<<B,T>>>(dev_a, dev_b, dev_c, dev_scalar);

	hipMemcpy(c,dev_c,N*sizeof(int),hipMemcpyDeviceToHost);

	for(int i=0;i<N;i++) {
		printf("(%d * %d) + %d = %d\n", *scalar, a[i], b[i], c[i]);
	}

	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

	return 0;
}
