#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <time.h>

#define N 16         // size of array

__global__ void calcsin(double *a, double *c) {
	int tid = blockIdx.x *  blockDim.x + threadIdx.x;
        if(tid < N){
        	c[tid] = sin(a[tid]);
        }
}

int main(int argc, char *argv[])  {
	int T = 10, B = 1;            // threads per block and blocks per grid
	double a[N], c[N];
	double *dev_a, *dev_c;

	printf("Size of array = %d\n", N);
	do {
		printf("Enter number of threads per block: ");
		scanf("%d",&T);
		printf("\nEnter nuumber of blocks per grid: ");
		scanf("%d",&B);
		if (T * B != N) printf("Error T x B != N, try again");
	} while (T * B != N);

	hipMalloc((void**)&dev_a,N * sizeof(double));
	hipMalloc((void**)&dev_c,N * sizeof(double));

	for(int i = 0; i < N; i++) {    // load array with some numbers
		a[i] = (double)i;
	}

	hipMemcpy(dev_a, a , N*sizeof(double),hipMemcpyHostToDevice);
	hipMemcpy(dev_c, c , N*sizeof(double),hipMemcpyHostToDevice);

	calcsin<<<B,T>>>(dev_a,dev_c);

	hipMemcpy(c,dev_c,N*sizeof(double),hipMemcpyDeviceToHost);


	for(int i = 0; i < N; i++) {
		printf("sin(%lf) = %lf\n",a[i], c[i]);
	}
	// clean up
	hipFree(dev_a);
	hipFree(dev_c);
	return 0;
}
